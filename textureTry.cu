#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> //checkCudaErrors()

#define cimg_use_tiff
#include <CImg.h>
using namespace cimg_library;

texture<float, hipTextureType2D, hipReadModeElementType> texRef1, texRef2;


// Simple transformation kernel
__global__ void scaleKernel(int width, int height, int scalex, 
                            int scaley, float* output1, float* output2)
{
// Calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x<width*scalex && y<height*scaley) {
    float u = ((float) x) / scalex / width;
    float v = ((float) y) / scaley / height;
    // Read from texture and write to global memory
    unsigned ind = y * width * scalex + x;
    output1[ind] = tex2D(texRef1, u, v);
    output2[ind] = tex2D(texRef2, u, v);
  }
}



int main(int argc, char *argv[])
{
  // load input file
  CImg<> indata(argv[1]);
  int nx = indata.width()/2;  // 2 because OTF is of float2 type
  int ny = indata.height();
  // split real and imag parts into two CImg arrays:
  CImg<> realpart(nx, ny), imagpart(nx, ny);

#pragma omp parallel for  
  cimg_forXY(realpart, x, y) {
    realpart(x, y) = indata(2*x  , y);
    imagpart(x, y) = indata(2*x+1, y);
  }
  

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  hipArray* cuArray1, *cuArray2;
  hipMallocArray(&cuArray1, &channelDesc, nx, ny);
  hipMallocArray(&cuArray2, &channelDesc, nx, ny);

  // Copy to device memory
  checkCudaErrors(hipMemcpyToArray(cuArray1, 0, 0, realpart.data(),
                                    realpart.size() * sizeof(float),
                                    hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpyToArray(cuArray2, 0, 0, imagpart.data(),
                                    imagpart.size() * sizeof(float),
                                    hipMemcpyHostToDevice));

  // hipMemcpy(realpart.data(), cuArray1,
  //            nx * ny * sizeof(float),
  //            hipMemcpyDeviceToHost);
  // hipMemcpy(imagpart.data(), cuArray2,
  //            nx * ny * sizeof(float),
  //            hipMemcpyDeviceToHost);
  // realpart.display();
  // imagpart.display();

  // Set texture reference parameters
  texRef1.addressMode[0] = hipAddressModeClamp;
  texRef1.addressMode[1] = hipAddressModeClamp;
  texRef1.filterMode = hipFilterModeLinear;
  texRef1.normalized = true;  // wonder what "false" would do here
  texRef2.addressMode[0] = hipAddressModeClamp;
  texRef2.addressMode[1] = hipAddressModeClamp;
  texRef2.filterMode = hipFilterModeLinear;
  texRef2.normalized = true;
  // Bind the array to the texture reference
  checkCudaErrors(hipBindTextureToArray(texRef1, cuArray1, channelDesc));
  checkCudaErrors(hipBindTextureToArray(texRef2, cuArray2, channelDesc));

  // Allocate result of scaling in device memory
  float* output1, *output2;
  int scalex=2, scaley=2;
  checkCudaErrors(hipMalloc(&output1, nx * scalex * ny * scaley * sizeof(float)));
  checkCudaErrors(hipMalloc(&output2, nx * scalex * ny * scaley * sizeof(float)));

  // Invoke kernel
  dim3 dimBlock(16, 16);
  dim3 dimGrid( (int) (ceil( ((float) nx*scalex) / dimBlock.x)),
                (int) (ceil( ((float) ny*scaley) / dimBlock.y)) );
  scaleKernel<<<dimGrid, dimBlock>>>(nx, ny, scalex, scaley, output1, output2);

  // Copy outputs back into host memory
  CImg<> realpartScaled(nx*scalex, ny*scaley), imagpartScaled(nx*scalex, ny*scaley);

  checkCudaErrors(hipMemcpy(realpartScaled.data(), output1,
                             nx * scalex * ny * scaley * sizeof(float),
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(imagpartScaled.data(), output2,
                             nx * scalex * ny * scaley * sizeof(float),
                             hipMemcpyDeviceToHost));
 
  realpartScaled.display();
  imagpartScaled.display();

  CImg<> combined(nx*scalex*2, ny*scaley);
#pragma omp parallel for  
  cimg_forXY(realpartScaled, x, y) {
    combined(2*x  , y) = realpartScaled(x, y);
    combined(2*x+1, y) = imagpartScaled(x, y);
  }
  
  combined.save("scaledOTF.tif");

  // Free device memory
  hipFreeArray(cuArray1);
  hipFreeArray(cuArray2);

  hipFree(output1);
  hipFree(output2);
  return 0;
}
