#include "hip/hip_runtime.h"
#include "cutilSafeCall.h"

#include <CPUBuffer.h>
#include <GPUBuffer.h>
#include <hipfft/hipfft.h>

__constant__ int const_nx;
__constant__ int const_ny;
__constant__ int const_nz;
__constant__ unsigned const_nxyz;
__constant__ int const_nrotf;
__constant__ int const_nzotf;

__constant__ float const_kxscale;
__constant__ float const_kyscale;
__constant__ float const_kzscale;
__constant__ float const_eps;
__constant__ hipFloatComplex const_otf[7680]; // 60 kB should be enough for an OTF array??

__global__ void filter_kernel(hipFloatComplex *devImg, hipFloatComplex *devOTF, int size, bool bConj);
__global__ void scale_kernel(float * img, double factor);
__global__ void LRcore_kernel(float * img1, float * img2);
__global__ void currEstimate_kernel(float * img1, float * img2, float * img3);
__global__ void currPrevDiff_kernel(float * img1, float * img2, float * img3);
__global__ void innerProduct_kernel(float * img1, float * img2,
                                    double * intRes1); //, double * intRes2);
__global__ void updatePrediction_kernel(float * Y_k, float * X_k, float *X_km1, float lambda);
__global__ void summation_kernel(float * img, double * intRes, int n);
__global__ void sumAboveThresh_kernel(float * img, double * intRes, unsigned * counter, float thresh, int n);

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
// (Copied from reduction_kernel.cu of CUDA samples)
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};


texture<float, hipTextureType2D, hipReadModeElementType> texRef1, texRef2;
hipArray* d_realpart, *d_imagpart;  // used for OTF texture


__host__ void transferConstants(int nx, int ny, int nz, int nrotf, int nzotf,
                                float kxscale, float kyscale, float kzscale,
                                float eps, float * h_otf)
{
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nx), &nx, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_ny), &ny, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nz), &nz, sizeof(int)));
  unsigned int nxyz = nx*ny*nz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nxyz), &nxyz, sizeof(unsigned int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nrotf), &nrotf, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_nzotf), &nzotf, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_kxscale), &kxscale, sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_kyscale), &kyscale, sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_kzscale), &kzscale, sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_eps), &eps, sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_otf), h_otf, nrotf*nzotf*2*sizeof(float)));
}

__host__ void prepareOTFtexture(float * realpart, float * imagpart, int nx, int ny)
{
  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  hipMallocArray(&d_realpart, &channelDesc, nx, ny);
  hipMallocArray(&d_imagpart, &channelDesc, nx, ny);

  // Copy to device memory
  hipMemcpyToArray(d_realpart, 0, 0, realpart,
                    nx * ny * sizeof(float),
                    hipMemcpyHostToDevice);
  hipMemcpyToArray(d_imagpart, 0, 0, imagpart,
                    nx * ny * sizeof(float),
                    hipMemcpyHostToDevice);

  // Set texture reference parameters
  texRef1.addressMode[0] = hipAddressModeClamp;
  texRef1.addressMode[1] = hipAddressModeClamp;
  texRef1.filterMode = hipFilterModeLinear;
  texRef1.normalized = true;
  texRef2.addressMode[0] = hipAddressModeClamp;
  texRef2.addressMode[1] = hipAddressModeClamp;
  texRef2.filterMode = hipFilterModeLinear;
  texRef2.normalized = true;
  // Bind the arrays to the texture reference
  hipBindTextureToArray(texRef1, d_realpart, channelDesc);
  hipBindTextureToArray(texRef2, d_imagpart, channelDesc);
}

__global__ void bgsubtr_kernel(float * img, int size, float background)
{
  int ind = blockIdx.x * blockDim.x + threadIdx.x;

  if (ind < size) {
    img[ind] -= background;
    img[ind] = img[ind] > 0 ? img[ind] : 0;
  }
}

__host__ void backgroundSubtraction_GPU(GPUBuffer &img, int nx, int ny, int nz, float background)
{
  int nThreads = 1024;
  int NXblock = (int) ceil( nx*ny*nz /(float) nThreads );
  dim3 grid(NXblock, 1, 1);
  dim3 block(nThreads, 1, 1);

  bgsubtr_kernel<<<grid, block>>>((float *) img.getPtr(), nx*ny*nz, background);
#ifndef NDEBUG
  std::cout<< "backgroundSubtraction_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

__host__ void filterGPU(GPUBuffer &img, int nx, int ny, int nz,
                        // GPUBuffer &otf,
                        hipfftHandle & rfftplan, hipfftHandle & rfftplanInv,
                        GPUBuffer &fftBuf,
                        GPUBuffer &otfArray, bool bConj)
// "img" is of dimension (nx, ny, nz) and of float type
// "otf" is of dimension (const_nzotf, const_nrotf) and of complex type
{
  hipfftResult cuFFTErr = hipfftExecR2C(rfftplan, (hipfftReal *) img.getPtr(),
                                      (hipFloatComplex *) fftBuf.getPtr());

  if (cuFFTErr != HIPFFT_SUCCESS) {
    std::cout << "Line:" << __LINE__ << std::endl;
    throw std::runtime_error("cufft failed.");
  }
  //
  // KERNEL 1
  //
  int nThreads = 1024;
  int arraySize = nz * ny * (nx/2+1);
  int NXblock = (int) ceil( arraySize / (float) nThreads );
  dim3 grid(NXblock);
  dim3 block(nThreads);

  filter_kernel<<<grid, block>>>((hipFloatComplex*) fftBuf.getPtr(),
                                 (hipFloatComplex*) otfArray.getPtr(),
                                 arraySize, bConj);

  cuFFTErr = hipfftExecC2R(rfftplanInv, (hipFloatComplex*)fftBuf.getPtr(), (hipfftReal *) img.getPtr());

  if (cuFFTErr != HIPFFT_SUCCESS) {
    std::cout << "Line:" << __LINE__ ;
    throw std::runtime_error("cufft failed.");
  }

  //
  // Rescale KERNEL
  //
  nThreads = 1024;
  NXblock = (int) ceil( ((float)(nx*ny*nz)) / nThreads );
  scale_kernel<<<NXblock, nThreads>>>((float *) img.getPtr(), 1./(nx*ny*nz));
}

__device__ hipFloatComplex dev_otfinterpolate(// hipFloatComplex * d_otf, 
                                             float kx, float ky, float kz)
  /* (kx, ky, kz) is Fourier space coords with origin at kx=ky=kz=0 and going  betwen -nx(or ny,nz)/2 and +nx(or ny,nz)/2 */
{
  float krindex = sqrt(kx*kx + ky*ky);
  float kzindex = (kz<0 ? kz+const_nzotf : kz);

  hipFloatComplex otfval = make_hipFloatComplex(0.f, 0.f);

  if (krindex < const_nrotf-1 && kzindex < const_nzotf) {
  // This should be rewritten using Textures for the interpolation. It will be much easier and faster!
    int irindex, izindex, indices[2][2];
    float ar, az;

    irindex = floor(krindex);
    izindex = floor(kzindex);

    ar = krindex - irindex;
    az = kzindex - izindex;  // az is always 0 for 2D case, and it'll just become a 1D interp

    if (izindex == const_nzotf-1) {
      indices[0][0] = irindex*const_nzotf+izindex;
      indices[0][1] = irindex*const_nzotf;
      indices[1][0] = (irindex+1)*const_nzotf+izindex;
      indices[1][1] = (irindex+1)*const_nzotf;
    }
    else {
      indices[0][0] = irindex*const_nzotf+izindex;
      indices[0][1] = irindex*const_nzotf+(izindex+1);
      indices[1][0] = (irindex+1)*const_nzotf+izindex;
      indices[1][1] = (irindex+1)*const_nzotf+(izindex+1);
    }
    otfval.x = (1-ar)*(const_otf[indices[0][0]].x*(1-az) + const_otf[indices[0][1]].x*az) +
      ar*(const_otf[indices[1][0]].x*(1-az) + const_otf[indices[1][1]].x*az);
    otfval.y = (1-ar)*(const_otf[indices[0][0]].y*(1-az) + const_otf[indices[0][1]].y*az) +
      ar*(const_otf[indices[1][0]].y*(1-az) + const_otf[indices[1][1]].y*az);
  }

  // float krindex = sqrt(kx*kx + ky*ky) / const_nrotf;
  // float kzindex = (kz<0 ? kz+const_nzotf : kz) / const_nzotf;

  // hipFloatComplex otfval;

  // otfval.x = tex2D(texRef1, kzindex, krindex);
  // otfval.y = tex2D(texRef2, kzindex, krindex);
  return otfval;
}

__global__ void filter_kernel(hipFloatComplex *devImg, hipFloatComplex *devOTF, int size, bool bConj)
{
  int ind = blockIdx.x * blockDim.x + threadIdx.x;

  if ( ind < size ) {
    hipFloatComplex otf_val = devOTF[ind];
    if (bConj)
      otf_val.y *= -1;
    devImg[ind] = hipCmulf(otf_val, devImg[ind]);
  }
}


__global__ void makeOTFarray_kernel(hipFloatComplex *result)
{
  int kx = blockIdx.x * blockDim.x + threadIdx.x;
  int ky = blockIdx.y > const_ny/2 ? blockIdx.y - const_ny : blockIdx.y;
  int kz = blockIdx.z > const_nz/2 ? blockIdx.z - const_nz : blockIdx.z;

  if (kx < const_nx/2+1) {
    hipFloatComplex otf_val = dev_otfinterpolate(kx*const_kxscale, ky*const_kyscale, kz*const_kzscale);
    unsigned ind = blockIdx.z * (const_nx/2+1) * const_ny  + blockIdx.y * (const_nx/2+1) + kx;
    result[ind].x = otf_val.x;
    result[ind].y = otf_val.y;
  }
}

__host__ void makeOTFarray(GPUBuffer &otfarray, int nx, int ny, int nz)
{
  unsigned nThreads=128;
  dim3 block(nThreads, 1, 1);
  unsigned blockNx = (int) ceil( (nx/2+1) / (float) nThreads );
  dim3 grid(blockNx, ny, nz);

  makeOTFarray_kernel<<<grid, block>>>( (hipFloatComplex *) otfarray.getPtr());
#ifndef NDEBUG
  std::cout<< "makeOTFarray(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

__global__ void scale_kernel(float * img, double factor)
{
  unsigned ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < const_nxyz)
    img[ind] *= factor;
}


__host__ void calcLRcore(GPUBuffer &reblurred, GPUBuffer &raw, int nx, int ny, int nz)
// calculate raw image divided by reblurred, a key step in R-L;
// Both input, "reblurred" and "raw", are of dimension (nx, ny, nz) and of floating type;
// "reblurred" is updated upon return.
{
  int nThreads = 1024;
  int NXblock = (int) ceil( ((float) (nx*ny*nz)) /nThreads );
  dim3 grid(NXblock, 1, 1);
  dim3 block(nThreads, 1, 1);

  LRcore_kernel<<<grid, block>>>((float *) reblurred.getPtr(), (float *) raw.getPtr());
}

__global__ void LRcore_kernel(float * img1, float * img2)
// Calculate img2/img1; results returned in img1
{
  int ind = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (ind < const_nxyz) {
    img1[ind] = img1[ind] > const_eps ? img1[ind] : const_eps;
    img1[ind] = img2[ind] / img1[ind];
  }
}

__host__ void updateCurrEstimate(GPUBuffer &X_k, GPUBuffer &CC, GPUBuffer &Y_k,
                                 int nx, int ny, int nz)
// calculate updated current estimate: Y_k * CC plus positivity constraint
// All inputs are of dimension (nx+2, ny, nz) and of floating type;
// "X_k" is updated upon return.
{
  int nThreads = 1024;
  int NXblock = (int) ceil( ((float) (nx*ny*nz)) / nThreads );
  dim3 grid(NXblock, 1, 1);
  dim3 block(nThreads, 1, 1);

  currEstimate_kernel<<<grid, block>>>((float *) X_k.getPtr(),
                                       (float *) CC.getPtr(),
                                       (float *) Y_k.getPtr());
}

__global__ void currEstimate_kernel(float * img1, float * img2, float * img3)
{
  int ind = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (ind < const_nxyz) {
    img1[ind] = img2[ind] * img3[ind];
    img1[ind] = img1[ind] > 0 ? img1[ind] : 0;
  }
}

__host__ void calcCurrPrevDiff(GPUBuffer &X_k, GPUBuffer &Y_k, GPUBuffer &G_kminus1,
                               int nx, int ny, int nz)
// calculate X_k - Y_k and assign the result to G_kminus1;
// All inputs are of dimension (nx+2, ny, nz) and of floating type;
// "X_k" is updated upon return.
{
  int nThreads = 1024; //128;
  int NXblock = (int) ceil( ((float) (nx*ny*nz)) / nThreads );
  dim3 grid(NXblock, 1, 1);
  dim3 block(nThreads, 1, 1);

  currPrevDiff_kernel<<<grid, block>>>((float *) X_k.getPtr(),
                                       (float *) Y_k.getPtr(),
                                       (float *) G_kminus1.getPtr());
}

__global__ void currPrevDiff_kernel(float * img1, float * img2, float * img3)
{
  // compute x, y, z indices based on block and thread indices
  int ind = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (ind < const_nxyz)
    img3[ind] = img1[ind] - img2[ind];
}

__host__ double calcAccelFactor(GPUBuffer &G_km1, GPUBuffer &G_km2,
                                int nx, int ny, int nz, float eps)
// (G_km1 dot G_km2) / (G_km2 dot G_km2)
// All inputs are of dimension (nx, ny, nz) and of floating type;
{
  int nThreads = 1024; // Maximum number of threads per block for C2070, M2090, or Quadro 4000
  int nBlocks = (int) ceil( ((float) (nx*ny*nz)) / nThreads/2 );

  // Used for holding partial reduction results; one for each thread block:
  GPUBuffer devBuf1(nBlocks * sizeof(double) * 2, 0);
  // First nBlocks: numerator; second nBlocks: denominator

  unsigned smemSize = nThreads * sizeof(double) * 2;
  innerProduct_kernel<<<nBlocks, nThreads, smemSize>>>((float *) G_km1.getPtr(),
                                                       (float *) G_km2.getPtr(),
                                                       (double *) devBuf1.getPtr());

  CPUBuffer h_numer_denom(devBuf1);

  double numerator=0, denom=0;
  double *ptr = (double *) h_numer_denom.getPtr();
  for (int i=0; i<nBlocks; i++) {
    numerator += *ptr;
    denom += *(ptr + nBlocks);
    ptr++;
  }

  return numerator / (denom + eps);
}

__global__ void innerProduct_kernel(float * img1, float * img2,
                                    double * intRes1)
// Using reduction to implement two inner products (img1.dot.img2 and img2.dot.img2)
// Copied from CUDA "reduction" sample code reduce4()
{
  double *sdata = SharedMemory<double>();
  // shared memory; even-numbered indices for img1.dot.img2;
  // odd-numbered indices for img2.dot.img2

  unsigned tid = threadIdx.x;
  unsigned ind = blockIdx.x * blockDim.x*2 + threadIdx.x;

  double mySum1=0, mySum2=0;
  if (ind< const_nxyz) {
    mySum1 = img1[ind] * img2[ind];
    mySum2 = img2[ind] * img2[ind];
  }

  unsigned indPlusBlockDim = ind + blockDim.x;
  if (indPlusBlockDim < const_nxyz) {
    mySum1 += img1[indPlusBlockDim] * img2[indPlusBlockDim];
    mySum2 += img2[indPlusBlockDim] * img2[indPlusBlockDim];
  }

  sdata[2*tid] = mySum1;
  sdata[2*tid + 1] = mySum2;
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
      sdata[2*tid] += sdata[2*(tid + s)];
      sdata[2*tid +1] += sdata[2*(tid + s) +1];
    }

    __syncthreads();
  }

  if (tid < 32) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile double *smem1 = sdata;

    // Assuming blockSize is > 64:
    smem1[2*tid] += smem1[2*(tid + 32)];
    smem1[2*tid] += smem1[2*(tid + 16)];
    smem1[2*tid] += smem1[2*(tid +  8)];
    smem1[2*tid] += smem1[2*(tid +  4)];
    smem1[2*tid] += smem1[2*(tid +  2)];
    smem1[2*tid] += smem1[2*(tid +  1)];
    smem1[2*tid+1] += smem1[2*(tid + 32)+1];
    smem1[2*tid+1] += smem1[2*(tid + 16)+1];
    smem1[2*tid+1] += smem1[2*(tid +  8)+1];
    smem1[2*tid+1] += smem1[2*(tid +  4)+1];
    smem1[2*tid+1] += smem1[2*(tid +  2)+1];
    smem1[2*tid+1] += smem1[2*(tid +  1)+1];
  }
  // write result for this block to global mem
  if (tid == 0) {
    intRes1[blockIdx.x] = sdata[0];
    intRes1[blockIdx.x + gridDim.x] = sdata[1];
  }
}

__host__ void updatePrediction(GPUBuffer &Y_k, GPUBuffer &X_k, GPUBuffer &X_kminus1,
                               double lambda, int nx, int ny, int nz)
{
  // Y_k = X_k + lambda * (X_k - X_kminus1)
  int nxyz = nx*ny*nz;
  int nThreads = 1024; // Maximum number of threads per block for C2070, M20990, or Quadro 4000
  int nBlocks = (int) ceil( ((float) nxyz) / nThreads );

  updatePrediction_kernel<<<nBlocks, nThreads>>>((float *) Y_k.getPtr(),
                                                 (float *) X_k.getPtr(),
                                                 (float *) X_kminus1.getPtr(),
                                                 lambda);
}

__global__ void updatePrediction_kernel(float * Y_k, float * X_k, float *X_km1, float lambda)
{
  unsigned ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < const_nxyz) {
    Y_k[ind] = X_k[ind] + lambda * (X_k[ind] - X_km1[ind]);
    Y_k[ind] = (Y_k[ind] > 0) ? Y_k[ind] : 0;
  }
}

__host__ double meanAboveBackground_GPU(GPUBuffer &img, int nx, int ny, int nz)
{
  unsigned nThreads = 1024;
  unsigned nBlocks = (unsigned) ceil( nx*ny*nz /(float) nThreads/2 );
  unsigned smemSize = nThreads * sizeof(double);

  // used for holding intermediate reduction results; one for each thread block
  GPUBuffer d_intres(nBlocks * sizeof(double), 0);

  summation_kernel<<<nBlocks, nThreads, smemSize>>>((float *) img.getPtr(),
                                                    (double *) d_intres.getPtr(), nx*ny*nz);
  // download intermediate results to host:
  CPUBuffer intRes(d_intres);
  double sum=0;
  double *p=(double *)intRes.getPtr();
  for (int i=0; i<nBlocks; i++)
    sum += *p++;

  float mean = sum/(nx*ny*nz);

  GPUBuffer d_counter(nBlocks * sizeof(unsigned), 0);
  smemSize = nThreads * (sizeof(double) + sizeof(unsigned));
  sumAboveThresh_kernel<<<nBlocks, nThreads, smemSize>>>((float *) img.getPtr(),
                                                         (double *) d_intres.getPtr(),
                                                         (unsigned *) d_counter.getPtr(),
                                                         mean, nx*ny*nz);
  
  // download intermediate results to host:
  CPUBuffer counter(d_counter);
  intRes = d_intres;
  sum=0;
  unsigned count = 0;
  p=(double *)intRes.getPtr();
  unsigned *pc = (unsigned *) counter.getPtr();
  for (int i=0; i<nBlocks; i++) {
    sum += *p++;
    count += *pc++;
  }

#ifndef NDEBUG
  printf("mean=%f, sum=%lf, count=%d\n", mean, sum, count);
#endif
  return sum/count;
}

__global__ void summation_kernel(float * img, double * intRes, int n)
// Copied from CUDA "reduction" sample code reduce4()
{
  double *sdata = SharedMemory<double>();

  unsigned tid = threadIdx.x;
  unsigned ind = blockIdx.x * blockDim.x*2 + threadIdx.x;

  double mySum= (ind < n) ? img[ind] : 0;

  if (ind + blockDim.x < n)
    mySum += img[ind + blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid < 32) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile double *smem = sdata;

    // Assuming blockSize is > 64:
    smem[tid] += smem[(tid + 32)];
    smem[tid] += smem[(tid + 16)];
    smem[tid] += smem[(tid +  8)];
    smem[tid] += smem[(tid +  4)];
    smem[tid] += smem[(tid +  2)];
    smem[tid] += smem[(tid +  1)];
  }
  // write result for this block to global mem
  if (tid == 0)
    intRes[blockIdx.x] = sdata[0];
}


__global__ void sumAboveThresh_kernel(float * img, double * intRes, unsigned * counter, float thresh, int n)
// Adapted from CUDA "reduction" sample code reduce4()
{
// Size of shared memory allocated is nThreads * (sizeof(double) + sizeof(unsigned))
// The first nThreads * sizeof(double) bytes are used for image intensity sum;
// the next nThreads * sizeof(unsigned) bytes are for counting pixels whose intensity is > thresh
  double *sdata = SharedMemory<double>();
  unsigned *count = (unsigned *) (sdata + blockDim.x);

  unsigned tid = threadIdx.x;
  unsigned ind = blockIdx.x * blockDim.x*2 + threadIdx.x;

  double mySum= 0;
  unsigned myCount = 0;
  if (ind < n && img[ind] > thresh) {
    mySum = img[ind]; 
    myCount ++;
  }

  unsigned ind2 = ind + blockDim.x;
  if (ind2 < n && img[ind2] > thresh) {
    mySum += img[ind2];
    myCount ++;
  }

  sdata[tid] = mySum;
  count[tid] = myCount;
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
      count[tid] += count[tid + s];
    }
    __syncthreads();
  }

  if (tid < 32) {
    volatile double *smem = sdata;
    volatile unsigned *cmem = count;

    smem[tid] += smem[(tid + 32)];
    smem[tid] += smem[(tid + 16)];
    smem[tid] += smem[(tid +  8)];
    smem[tid] += smem[(tid +  4)];
    smem[tid] += smem[(tid +  2)];
    smem[tid] += smem[(tid +  1)];
    cmem[tid] += cmem[(tid + 32)];
    cmem[tid] += cmem[(tid + 16)];
    cmem[tid] += cmem[(tid +  8)];
    cmem[tid] += cmem[(tid +  4)];
    cmem[tid] += cmem[(tid +  2)];
    cmem[tid] += cmem[(tid +  1)];
  }
  // write result for this block to global mem
  if (tid == 0) {
    intRes[blockIdx.x] = sdata[0];
    counter[blockIdx.x] = count[0];
  }
}

__host__ void rescale_GPU(GPUBuffer &img, int nx, int ny, int nz, float scale)
{
  unsigned nThreads = 1024;
  unsigned nBlocks = (unsigned) ceil( nx*ny*nz / (float) nThreads );
  scale_kernel<<<nBlocks, nThreads>>>((float *) img.getPtr(), scale);
#ifndef NDEBUG
  std::cout<< "rescale_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}
